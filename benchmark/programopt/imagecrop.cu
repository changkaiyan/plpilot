
#include <hip/hip_runtime.h>
__global__ void crop_kernel(unsigned char *input, unsigned char *output, 
                            int input_rows, int input_cols,
                            int output_rows, int output_cols,
                            int row_start, int col_start) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < output_rows && col < output_cols) {
        int input_row = row + row_start;
        int input_col = col + col_start;
        output[row * output_cols + col] = input[input_row * input_cols + input_col];
    } 
}