#include <hip/hip_runtime.h>

__global__ void kernel() {
    // Grid has only 100 blocks, but each block has 1024 threads!
    // This leads to thread overflow as 104,400 threads are launched
    printf("Thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

int main() {
    const int num_blocks = 100;
    const int threads_per_block = 1024;
    const int num_threads = num_blocks * threads_per_block;
    
    kernel<<<num_blocks, threads_per_block>>>(); 
    
    hipDeviceSynchronize(); 
}