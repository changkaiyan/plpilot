#include <hip/hip_runtime.h>

__global__ void kernel1() {
    printf("Kernel 1 launching\n");
}

__global__ void kernel2() {
    printf("Kernel 2 launching\n");
}

int main() {
    kernel1<<<1, 1>>>();
    
    // Launch kernel2 without synchronizing from kernel1 - error!
    kernel2<<<1, 1>>>();
    
    hipDeviceSynchronize(); 
}